#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define STREAM_NUM  4
__constant__ float kernel_mask[1 * 7 * 7 * 4 * 16];

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, 
    const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int W_grid = ceil((Width_out*1.0)/TILE_WIDTH);
    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) kernel_mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    
    int n, m, h, w, c;  // image_index, map_inedx, specific_height, specific_width, channel
    n = blockIdx.x;
    m = blockIdx.y;
    h = (blockIdx.z / W_grid) * TILE_WIDTH + threadIdx.y;
    w = (blockIdx.z % W_grid) * TILE_WIDTH + threadIdx.x;
    float acc = 0;
    if(h < Height_out && w < Width_out){
        for(c = 0; c < Channel; ++c){
            for(int p = 0; p < K; ++p){     // for loop, the mask K x K
                for(int q = 0; q < K; ++q){
                    acc += in_4d(n, c, h+p, w+q) * mask_4d(m,c,p,q);
                }
            }
        }
        out_4d(n,m,h,w) = acc;
    }
    
    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    int H_out = Height - K + 1; // compute the output height
    int W_out = Width - K + 1;  // compute the input width
    hipMalloc((void**)device_output_ptr, Batch * Map_out * H_out * W_out * sizeof(float));   // using the output height and width
    hipMalloc((void**)device_input_ptr, Batch * Channel * Height * Width * sizeof(float));   // using the input height and width       
    hipMemcpy(*device_input_ptr, host_input, Batch * Channel * Height * Width * sizeof(float), hipMemcpyHostToDevice); // copy host input to device input    
    hipMemcpyToSymbol(HIP_SYMBOL(kernel_mask), host_mask, 1 * 7 * 7 * 4 * 16*sizeof(float)); // copy to the constant memory


    // defined the grids required for the output
    int H_grid = ceil((H_out*1.0)/TILE_WIDTH);  // height grids 
    int W_grid = ceil((W_out*1.0)/TILE_WIDTH);  // width grids
    int Z = H_grid * W_grid;                    // h x w requirements for the image
    const int segSize = 25; // define seg size
    
    hipStream_t streams[STREAM_NUM];
    for(int i = 0; i < STREAM_NUM; ++i){    // create streams
        hipStreamCreate(&streams[i]);
    } 
    
    int input_CHW = Channel * Height * Width;
    int output_MHW = Map_out * H_out * W_out;
    int in_copy_size = segSize * input_CHW;
    int out_copy_size = segSize * output_MHW;
    
    dim3 DimBlocks(TILE_WIDTH,TILE_WIDTH, 1); // the un-optimized one
    dim3 DimGrids(segSize, Map_out, Z);              // grid dimension

    for (int i = 0; i < Batch; i += (STREAM_NUM * segSize)){

        // input offsets
        int offset0 = (i + 0 * segSize) * input_CHW;
        int offset1 = (i + 1 * segSize) * input_CHW;
        int offset2 = (i + 2 * segSize) * input_CHW;
        int offset3 = (i + 3 * segSize) * input_CHW;

        // output offset
        int out_offset0 = (i + 0 * segSize) * output_MHW;
        int out_offset1 = (i + 1 * segSize) * output_MHW;
        int out_offset2 = (i + 2 * segSize) * output_MHW;
        int out_offset3 = (i + 3 * segSize) * output_MHW;

        // async cpy input, host to device
        hipMemcpyAsync(*device_input_ptr + offset0, host_input + offset0, in_copy_size * sizeof(float), hipMemcpyHostToDevice, streams[0]);
        hipMemcpyAsync(*device_input_ptr + offset1, host_input + offset1, in_copy_size * sizeof(float), hipMemcpyHostToDevice, streams[1]);
        hipMemcpyAsync(*device_input_ptr + offset2, host_input + offset2, in_copy_size * sizeof(float), hipMemcpyHostToDevice, streams[2]);
        hipMemcpyAsync(*device_input_ptr + offset3, host_input + offset3, in_copy_size * sizeof(float), hipMemcpyHostToDevice, streams[3]);

        // stream kernel calls
        conv_forward_kernel<<<DimGrids, DimBlocks, 0, streams[0]>>>(*device_output_ptr + out_offset0, *device_input_ptr + offset0, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        conv_forward_kernel<<<DimGrids, DimBlocks, 0, streams[1]>>>(*device_output_ptr + out_offset1, *device_input_ptr + offset1, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        conv_forward_kernel<<<DimGrids, DimBlocks, 0, streams[2]>>>(*device_output_ptr + out_offset2, *device_input_ptr + offset2, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        conv_forward_kernel<<<DimGrids, DimBlocks, 0, streams[3]>>>(*device_output_ptr + out_offset3, *device_input_ptr + offset3, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);

        // async cpy output, device to host
        hipMemcpyAsync((void*)(host_output + out_offset0), *device_output_ptr + out_offset0, out_copy_size * sizeof(float), hipMemcpyDeviceToHost, streams[0]);
        hipMemcpyAsync((void*)(host_output + out_offset1), *device_output_ptr + out_offset1, out_copy_size * sizeof(float), hipMemcpyDeviceToHost, streams[1]);
        hipMemcpyAsync((void*)(host_output + out_offset2), *device_output_ptr + out_offset2, out_copy_size * sizeof(float), hipMemcpyDeviceToHost, streams[2]);
        hipMemcpyAsync((void*)(host_output + out_offset3), *device_output_ptr + out_offset3, out_copy_size * sizeof(float), hipMemcpyDeviceToHost, streams[3]);
    }

    hipFree(*device_input_ptr);
    hipFree(*device_output_ptr);
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    // int H_out = Height - K + 1;
    // int W_out = Width - K + 1;
    
    // // defined the grids required for the output
    // int H_grid = ceil((H_out*1.0)/TILE_WIDTH);  // height grids 
    // int W_grid = ceil((W_out*1.0)/TILE_WIDTH);  // width grids
    // int Z = H_grid * W_grid;                    // h x w requirements for the image


    return;
    // hipDeviceSynchronize();

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    // int H_out = Height - K + 1;
    // int W_out = Width - K + 1;    
    // Free device memory
    return;
}

__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
