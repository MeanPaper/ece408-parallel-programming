#include "hip/hip_runtime.h"
// MP 5.1 Reduction
// Given a list of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include <wb.h>

// the original one is 512
#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)
  
__global__ void total(float *input, float *output, int len) {
  //@@ Load a segment of the input vector into shared memory
  //@@ Traverse the reduction tree
  //@@ Write the computed sum of the block to the output vector at the
  //@@ correct index
  __shared__ float tileMem[BLOCK_SIZE*2];
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int bdim = blockDim.x;
  int start = bx * bdim * 2;
  if(start + tx < len){
    tileMem[tx] = input[start+tx]; 
  }
  else{
    tileMem[tx] = 0.0;
  }
  if(start + tx + bdim < len){
    tileMem[tx + bdim] = input[start + tx + bdim];
  }
  else{
    tileMem[tx + bdim] = 0.0;
  }
  
  for(int stride = bdim; stride > 1; stride /= 2){
    __syncthreads();
    if(tx < stride){
      tileMem[tx] += tileMem[tx+stride];
    }
  }
  if(tx == 0){  // have control divergence 
    output[bx] = tileMem[tx] + tileMem[tx+1];
  }
  // this one does not have control divergence; however, it might have a lot of loading overhead
  // output[bx] = tileMen[0]; 
}

int main(int argc, char **argv) {
  int ii;
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numInputElements;  // number of elements in the input list
  int numOutputElements; // number of elements in the output list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput =
      (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

  numOutputElements = (numInputElements - 1) / (BLOCK_SIZE << 1) + 1;
  hostOutput = (float *)malloc(numOutputElements * sizeof(float));

  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numInputElements);
  wbLog(TRACE, "The number of output elements in the input is ",
        numOutputElements);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceInput, numInputElements*sizeof(float));
  hipMalloc((void **)&deviceOutput, numInputElements*sizeof(float));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");

  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput, numInputElements*sizeof(float), hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");
  //@@ Initialize the grid and block dimensions here
  dim3 GridDim(ceil(numInputElements/(2.0 * BLOCK_SIZE)), 1,1);  // two thread do two job
  dim3 BlockDim(BLOCK_SIZE,1,1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  total<<<GridDim, BlockDim>>>(deviceInput, deviceOutput, numInputElements);
  
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, numOutputElements*sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  /***********************************************************************
   * Reduce output vector on the host
   * NOTE: One could also perform the reduction of the output vector
   * recursively and support any size input.
   * For simplicity, we do not require that for this lab!
   ***********************************************************************/
  for (ii = 1; ii < numOutputElements; ii++) {
    // printf("%d \n", hostOutput[ii]);
    hostOutput[0] += hostOutput[ii];
  }

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, 1);

  free(hostInput);
  free(hostOutput);

  return 0;
}
