#include "hip/hip_runtime.h"
#include <wb.h>

#define BLOCK_SIZE 512

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)


// out: the output array
// matColStart: JDS col pointers, same as the one in the lectures
// matCol: the specific column index with respect to the col pointers
// matRowPerm: the row index of data, the row indexes are sorted based on the # of data in the row in the original entry, sorted from largest to smallest

// matRows is very clear, and it is not mentioned in the slide, I will just use an example for it
// matRows: the number of rows used in the transposed JDS format by the original JDS format

// vec: the vector to be multiplied
// dim: the length of the vector

/**
 * for exmaple, a JDS format might look the following
 * 
 * // the original data are stored as the following
 * 2 4 1    row 2 
 * 3 1      row 0
 * 1 1      row 3
 *          row 1, has no data
 * 
 * // JDS transposed
 * // the index 0,1,2 is referring to the index of the output entry
 * // the mat row is basically the max number of second that each thread needs to iterate
 * // it also means that the # of data in each row in JDS format, required that # of row to stores in JDS transposed format
 * // columns used by the row = 4, then that row of data will required 4 rows in JDS transposed format
 * 
 * 2 3 1    matRow[0] = 3
 * 4 1 1    matRow[1] = 3
 * 1 1      matRow[2] = 2
*/
__global__ void spmvJDSKernel(float *out, int *matColStart, int *matCols,
                              int *matRowPerm, int *matRows,
                              float *matData, float *vec, int dim) {
  //@@ insert spmv kernel for jds format
  int row = blockDim.x * blockIdx.x + threadIdx.x;
  if(row < dim){
    float dotProduct = 0;
    int sec = 0;
    int counts = matRows[row]; // here we will get the number of rows used in JDS transposed format used by this "row" of data in the JDS format
    while(sec < counts){
      int index = matColStart[sec] + row; // here we want to find the matData element we are looking at
      dotProduct += matData[index] * vec[matCols[index]]; // compute the dot product
      ++sec;
    }
    // we are loading the information back based on the permutation, less say row = 0, but in Perm[0] is 2 
    // we actually use thread 0 to compute the data at index 2 of the output
    out[matRowPerm[row]] = dotProduct; 
  }
}

static void spmvJDS(float *out, int *matColStart, int *matCols,
                    int *matRowPerm, int *matRows, float *matData,
                    float *vec, int dim) {

  //@@ invoke spmv kernel for jds format
  dim3 DimBlock(BLOCK_SIZE,1,1);
  dim3 DimGrid(ceil(dim*1.0 / BLOCK_SIZE),1,1);
  spmvJDSKernel<<<DimGrid, DimBlock>>>(out, matColStart, matCols, matRowPerm, matRows, matData, vec, dim);
}

int main(int argc, char **argv) {
  wbArg_t args;
  int *hostCSRCols;
  int *hostCSRRows;
  float *hostCSRData;
  int *hostJDSColStart;
  int *hostJDSCols;
  int *hostJDSRowPerm;
  int *hostJDSRows;
  float *hostJDSData;
  float *hostVector;
  float *hostOutput;
  int *deviceJDSColStart;
  int *deviceJDSCols;
  int *deviceJDSRowPerm;
  int *deviceJDSRows;
  float *deviceJDSData;
  float *deviceVector;
  float *deviceOutput;
  int dim, ncols, nrows, ndata;
  int maxRowNNZ;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostCSRCols = (int *)wbImport(wbArg_getInputFile(args, 0), &ncols, "Integer");
  hostCSRRows = (int *)wbImport(wbArg_getInputFile(args, 1), &nrows, "Integer");
  hostCSRData = (float *)wbImport(wbArg_getInputFile(args, 2), &ndata, "Real");
  hostVector = (float *)wbImport(wbArg_getInputFile(args, 3), &dim, "Real");

  hostOutput = (float *)malloc(sizeof(float) * dim);

  wbTime_stop(Generic, "Importing data and creating memory on host");

  CSRToJDS(dim, hostCSRRows, hostCSRCols, hostCSRData, &hostJDSRowPerm, &hostJDSRows,
           &hostJDSColStart, &hostJDSCols, &hostJDSData);
  maxRowNNZ = hostJDSRows[0];

  wbTime_start(GPU, "Allocating GPU memory.");
  hipMalloc((void **)&deviceJDSColStart, sizeof(int) * maxRowNNZ);
  hipMalloc((void **)&deviceJDSCols, sizeof(int) * ndata);
  hipMalloc((void **)&deviceJDSRowPerm, sizeof(int) * dim);
  hipMalloc((void **)&deviceJDSRows, sizeof(int) * dim);
  hipMalloc((void **)&deviceJDSData, sizeof(float) * ndata);

  hipMalloc((void **)&deviceVector, sizeof(float) * dim);
  hipMalloc((void **)&deviceOutput, sizeof(float) * dim);
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  hipMemcpy(deviceJDSColStart, hostJDSColStart, sizeof(int) * maxRowNNZ,
             hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSCols, hostJDSCols, sizeof(int) * ndata, hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSRowPerm, hostJDSRowPerm, sizeof(int) * dim, hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSRows, hostJDSRows, sizeof(int) * dim, hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSData, hostJDSData, sizeof(float) * ndata, hipMemcpyHostToDevice);
  hipMemcpy(deviceVector, hostVector, sizeof(float) * dim, hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  wbTime_start(Compute, "Performing CUDA computation");
  spmvJDS(deviceOutput, deviceJDSColStart, deviceJDSCols, deviceJDSRowPerm, deviceJDSRows,
          deviceJDSData, deviceVector, dim);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  hipMemcpy(hostOutput, deviceOutput, sizeof(float) * dim, hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceVector);
  hipFree(deviceOutput);
  hipFree(deviceJDSColStart);
  hipFree(deviceJDSCols);
  hipFree(deviceJDSRowPerm);
  hipFree(deviceJDSRows);
  hipFree(deviceJDSData);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, dim);

  free(hostCSRCols);
  free(hostCSRRows);
  free(hostCSRData);
  free(hostVector);
  free(hostOutput);
  free(hostJDSColStart);
  free(hostJDSCols);
  free(hostJDSRowPerm);
  free(hostJDSRows);
  free(hostJDSData);

  return 0;
}
